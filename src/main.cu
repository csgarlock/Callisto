#include "hip/hip_runtime.h"
#include "activation.h"
#include "util.h"
#include "matrix.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <random>

template <typename Activation>
void test_activation(const int n) {
    std::cout << "Testing activation with n = " << n << std::endl;

    std::vector<float> host_in(n);
    std::vector<float> host_out_cpu(n);

    // Fill input with simple pattern
    for (int i = 0; i < n; ++i)
        host_in[i] = 0.0f;
        // host_in[i] = std::sin(i * 0.01f) * 5.0f;

    float *dev_vec = nullptr;
    CUDA_CHECK(hipMalloc(&dev_vec, n * sizeof(float)));
    CUDA_CHECK(hipMemcpy(dev_vec, host_in.data(), n * sizeof(float), hipMemcpyHostToDevice));

    // Launch config
    const int threads = 256;
    const int blocks = (n / 4 + threads - 1) / threads;

    // Timing with CUDA events
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    activation<Activation><<<blocks, threads>>>(dev_vec, n);
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));

    float elapsed_ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));

    // Copy back results
    std::vector<float> host_out_gpu(n);
    CUDA_CHECK(hipMemcpy(host_out_gpu.data(), dev_vec, n * sizeof(float), hipMemcpyDeviceToHost));

    // Compute reference on CPU
    for (int i = 0; i < n; ++i)
        host_out_cpu[i] = Activation::cpu(host_in[i]);

    // Validate
    float max_err = 0.0f;
    for (int i = 0; i < n; ++i) {
        max_err = std::max(max_err, std::abs(host_out_cpu[i] - host_out_gpu[i]));
    }
    // Bandwidth calculation
    double bytes = static_cast<double>(n) * sizeof(float) * 2; // read + write
    double gbps = bytes / (elapsed_ms / 1e3) / 1e9;

    std::cout << "Max error: " << max_err << std::endl;
    std::cout << "Elapsed: " << elapsed_ms << " ms" << std::endl;
    std::cout << "Throughput: " << gbps << " GB/s" << std::endl;

    CUDA_CHECK(hipFree(dev_vec));
}

void test_feed_forward(int m, int n) {
    std::cout << "Testing feed_forward with m=" << m << ", n=" << n << std::endl;

    // Host memory
    std::vector<float> h_input(n);
    std::vector<float> h_weights(m * n);
    std::vector<float> h_biases(m);
    std::vector<float> h_output_cpu(m);
    std::vector<float> h_output_gpu(m);

    // Initialize data
    std::mt19937 rng(42);
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for (int i = 0; i < n; ++i) h_input[i] = dist(rng);
    for (int i = 0; i < m * n; ++i) h_weights[i] = dist(rng);
    for (int i = 0; i < m; ++i) h_biases[i] = dist(rng);

    // CPU reference
    for (int row = 0; row < m; ++row) {
        float sum = h_biases[row];
        for (int col = 0; col < n; ++col)
            sum += h_weights[row * n + col] * h_input[col];
        h_output_cpu[row] = sum;
    }

    // Device memory
    float *d_input, *d_weights, *d_biases, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_weights, m * n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_biases, m * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, m * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), n * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, h_weights.data(), m * n * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_biases, h_biases.data(), m * sizeof(float), hipMemcpyHostToDevice));

    // Kernel configuration
    const int threads = 256;
    const int blocks = std::min(m, 128);  // up to 128 rows in parallel

    MatrixMultShape shape{n, m};

    // Timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    feed_forward<<<blocks, threads>>>(d_input, d_weights, d_biases, d_output, shape);
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));

    float elapsed_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));

    CUDA_CHECK(hipMemcpy(h_output_gpu.data(), d_output, m * sizeof(float), hipMemcpyDeviceToHost));

    // Validation
    float max_err = 0.0f;
    for (int i = 0; i < m; ++i)
        max_err = std::max(max_err, std::abs(h_output_cpu[i] - h_output_gpu[i]));

    // Performance metrics
    double flops = static_cast<double>(m) * n * 2.0; // mul + add per element
    double gflops = (flops / (elapsed_ms / 1e3)) / 1e9;

    std::cout << "Max error: " << max_err << std::endl;
    std::cout << "Elapsed: " << elapsed_ms << " ms" << std::endl;
    std::cout << "Performance: " << gflops << " GFLOP/s" << std::endl;

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_weights));
    CUDA_CHECK(hipFree(d_biases));
    CUDA_CHECK(hipFree(d_output));
}

int main() {
    // Example: 4096×4096 matvec (big enough for stable timing)
    test_feed_forward(4096, 4096);
    test_feed_forward(8192, 32768);
    test_feed_forward(32768, 8192);
    return 0;
}

// int main() {
//     int n = 1 << 26;  // 16 million elements (64 MB)
//     test_activation<ReLU>(n);
//     test_activation<LogisticSigmoid>(n);
//     return 0;
// }