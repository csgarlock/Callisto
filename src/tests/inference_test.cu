#include "hip/hip_runtime.h"
#include "inference_test.h"

#include "../kernels/linear.h"

void test_feed_forward(int m, int n) {
    std::cout << "Testing feed_forward with m=" << m << ", n=" << n << std::endl;

    // Host memory
    std::vector<float> h_input(n);
    std::vector<float> h_weights(m * n);
    std::vector<float> h_biases(m);
    std::vector<float> h_output_cpu(m);
    std::vector<float> h_output_gpu(m);

    // Initialize data
    std::mt19937 rng(42);
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for (int i = 0; i < n; ++i) h_input[i] = dist(rng);
    for (int i = 0; i < m * n; ++i) h_weights[i] = dist(rng);
    for (int i = 0; i < m; ++i) h_biases[i] = dist(rng);

    // CPU reference
    for (int row = 0; row < m; ++row) {
        float sum = h_biases[row];
        for (int col = 0; col < n; ++col)
            sum += h_weights[row * n + col] * h_input[col];
        h_output_cpu[row] = sum;
    }

    // Device memory
    float *d_input, *d_weights, *d_biases, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_weights, m * n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_biases, m * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, m * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), n * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, h_weights.data(), m * n * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_biases, h_biases.data(), m * sizeof(float), hipMemcpyHostToDevice));

    // Kernel configuration
    const int threads = 256;
    const int blocks = std::min(m / 32, 256);  // up to 256 tiles in parallel

    std::cout << "Launch Config. Blocks: " << blocks << ", Threads per Block: " << threads << std::endl; 

    MatrixMultShape shape{n, m};

    // Timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    linear_forward_mtm<<<blocks, threads>>>(d_input, d_weights, d_biases, d_output, shape);
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));

    float elapsed_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));

    CUDA_CHECK(hipMemcpy(h_output_gpu.data(), d_output, m * sizeof(float), hipMemcpyDeviceToHost));

    // Validation
    float max_err = 0.0f;
    for (int i = 0; i < m; ++i) {
        max_err = std::max(max_err, std::abs(h_output_cpu[i] - h_output_gpu[i]));
    }
    // Performance metrics
    double flops = static_cast<double>(m) * n * 2.0; // mul + add per element
    double gflops = (flops / (elapsed_ms / 1e3)) / 1e9;

    std::cout << "Max error: " << max_err << std::endl;
    std::cout << "Elapsed: " << elapsed_ms << " ms" << std::endl;
    std::cout << "Performance: " << gflops << " GFLOP/s" << std::endl;

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_weights));
    CUDA_CHECK(hipFree(d_biases));
    CUDA_CHECK(hipFree(d_output));
}

void test_feed_forward_batch(int m, int n, int batch_size) {
    std::cout << "Testing linear_forward_mtm_batch with m=" << m
              << ", n=" << n
              << ", batch_size=" << batch_size << std::endl;

    // Host memory
    std::vector<float> h_input(batch_size * n);
    std::vector<float> h_weights(m * n);
    std::vector<float> h_biases(m);
    std::vector<float> h_output_cpu(batch_size * m);
    std::vector<float> h_output_gpu(batch_size * m);

    // Initialize random data
    std::mt19937 rng(42);
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for (int i = 0; i < batch_size * n; ++i) h_input[i] = dist(rng);
    for (int i = 0; i < m * n; ++i) h_weights[i] = dist(rng);
    for (int i = 0; i < m; ++i) h_biases[i] = dist(rng);

    // ---- CPU Reference ----
    for (int b = 0; b < batch_size; ++b) {
        for (int row = 0; row < m; ++row) {
            float sum = h_biases[row];
            for (int col = 0; col < n; ++col) {
                sum += h_weights[row * n + col] * h_input[b * n + col];
            }
            h_output_cpu[b * m + row] = sum;
        }
    }

    // ---- Device memory ----
    float *d_input, *d_weights, *d_biases, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, batch_size * n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_weights, m * n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_biases, m * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, batch_size * m * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), batch_size * n * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, h_weights.data(), m * n * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_biases, h_biases.data(), m * sizeof(float), hipMemcpyHostToDevice));

    // ---- Kernel configuration ----
    const int threads = 256;
    dim3 blocks(
        std::min(m / 32, 32),   // tile rows
        std::min(batch_size / 32, 8) // tile depth (batch dimension)
    );

    std::cout << "Launch Config -> Blocks: (" << blocks.x << "," << blocks.y
              << "), Threads: " << threads << std::endl;

    MatrixMultShape shape{n, m, batch_size};

    // ---- Timing ----
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start));
    linear_forward_mtm_batch<Identity><<<blocks, threads>>>(d_input, d_weights, d_biases, d_output, shape);
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));

    float elapsed_ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));

    // ---- Copy back ----
    CUDA_CHECK(hipMemcpy(h_output_gpu.data(), d_output,
                          batch_size * m * sizeof(float),
                          hipMemcpyDeviceToHost));

    // ---- Validate ----
    float max_err = 0.0f;
    for (int i = 0; i < batch_size * m; ++i)
        max_err = std::max(max_err, std::abs(h_output_cpu[i] - h_output_gpu[i]));

    // ---- Performance ----
    double flops = static_cast<double>(batch_size) * m * n * 2.0; // mul + add per element
    double gflops = (flops / (elapsed_ms / 1e3)) / 1e9;

    std::cout << "Max error: " << max_err << std::endl;
    std::cout << "Elapsed: " << elapsed_ms << " ms" << std::endl;
    std::cout << "Throughput: " << gflops << " GFLOP/s" << std::endl;

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_weights));
    CUDA_CHECK(hipFree(d_biases));
    CUDA_CHECK(hipFree(d_output));
}
